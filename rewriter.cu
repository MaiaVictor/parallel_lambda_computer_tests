
#include <hip/hip_runtime.h>
__host__ __device__ void swap(int* a, int* b){
    int temp = *a;
    *a = *b;
    *b = temp;
}

__host__ __device__ void swap4(int* a, int* b){
    for (int i=0; i<4; ++i)
        swap(a+i, b+i);
};

__host__ __device__ void erase(int* a){
    for (int i=0; i<4; ++i)
        a[i] = 0;
};

// Rewrites 3 memory nodes based on interaction rules
__host__ __device__ void rewrite(int* a, int* b, int* c){
    int a0, a1, a2, a3,
        b0, b1, b2, b3,
        c0, c1, c2, c3,
        tmp;

    a0 = a[0], a1 = a[1], a2 = a[2], a3 = a[3];
    b0 = b[0], b1 = b[1], b2 = b[2], b3 = b[3];
    c0 = c[0], c1 = c[1], c2 = c[2], c3 = c[3];

    // Full block
    if (!(a0 || b0 || c0) || (a0 && b0 && c0))
        return;

    // Move
    if (!(a0 || b0) || (a0 && b0))
        tmp = b0, b0 = c0, c0 = tmp,
        tmp = b1, b1 = c1, c1 = tmp,
        tmp = b2, b2 = c2, c2 = tmp,
        tmp = b3, b3 = c3, c3 = tmp;

    // Duplicate
    if (a0 < 0 && b0 == 0)
        tmp = a1, a1 = a2, a2 = tmp,
        b0 = -a0, b1 = a3, a0 *= -1,
        b2 = a2 + (a2>0?2:-1),
        a3 = a2 + (a2>0?1:-2),
        b3 = a2 + (a2>0?3:-3);

    // Nothing to do
    if ((a0>0 && !b0 && c0>0)){

        // Passthrough
        if (   (a1 >  0 && c1  < 0)
            || (a1 >  0 && a0 == 1)
            || (c0 == 1 && c1  < 0)){
            tmp = a0, a0 = c0, c0 = tmp,
            tmp = a1, a1 = c1, c1 = tmp,
            tmp = a2, a2 = c2, c2 = tmp,
            tmp = a3, a3 = c3, c3 = tmp;
            a1 *= a1==-c1 || a1==-c2 || a1==-c3 ? -1 : 1;
            a2 *= a2==-c1 || a2==-c2 || a2==-c3 ? -1 : 1;
            a3 *= a3==-c1 || a3==-c2 || a3==-c3 ? -1 : 1;
            c1 *= c1== a1 || c1== a2 || c1== a3 ? -1 : 1;
            c2 *= c2== a1 || c2== a2 || c2== a3 ? -1 : 1;
            c3 *= c3== a1 || c3== a2 || c3== a3 ? -1 : 1;
        };

        // Substitute
        // TODO: clean this mess
        if (c0 == 1 && c1 == -a1){ a1 = c2; c0=c1=c2=c3=0; if (a2 == -a3 && a2 < 0) tmp = a2, a2 = a3, a3 = tmp; if (a1 == -a2) a0=a1=a2=a3=0; };
        if (c0 == 1 && c1 == -a2){ a2 = c2; c0=c1=c2=c3=0; if (a2 == -a3 && a2 < 0) tmp = a2, a2 = a3, a3 = tmp; if (a1 == -a2) a0=a1=a2=a3=0; };
        if (c0 == 1 && c1 == -a3){ a3 = c2; c0=c1=c2=c3=0; if (a2 == -a3 && a2 < 0) tmp = a2, a2 = a3, a3 = tmp; if (a1 == -a2) a0=a1=a2=a3=0; };
        if (a0 == 1 && a1 == -c1){ c1 = a2; a0=a1=a2=a3=0; if (c2 == -c3 && c2 < 0) tmp = c2, c2 = c3, c3 = tmp; if (c1 == -c2) c0=c1=c2=c3=0; };
        if (a0 == 1 && a1 == -c2){ c2 = a2; a0=a1=a2=a3=0; if (c2 == -c3 && c2 < 0) tmp = c2, c2 = c3, c3 = tmp; if (c1 == -c2) c0=c1=c2=c3=0; };
        if (a0 == 1 && a1 == -c3){ c3 = a2; a0=a1=a2=a3=0; if (c2 == -c3 && c2 < 0) tmp = c2, c2 = c3, c3 = tmp; if (c1 == -c2) c0=c1=c2=c3=0; };

        // React
        if (a0 > 1 && c0 > 1 && a1 == -c1){
            if (a0 == c0)
                a0 = 1, a1 = a2, a2 = c2,
                c0 = 1, c1 = a3, c2 = c3,
                a3 = 0, c3 = 0;
            else
                tmp = a0, a0 = c0, c0 = tmp,
                a0 *= -1,
                c0 *= -1,
                a1 = (a1*1103515245+12345)&0x7fffffff, // PRNG
                c1 = -a1;
        };
    }

    a[0] = a0, a[1] = a1, a[2] = a2, a[3] = a3;
    b[0] = b0, b[1] = b1, b[2] = b2, b[3] = b3;
    c[0] = c0, c[1] = c1, c[2] = c2, c[3] = c3;
};

__host__ __device__ void tick(int *mem, int total_nodes){
    for (int j=0; j<3; ++j)
        for (int i=j; i<total_nodes-2; i+=3)
            rewrite(mem+i*4, mem+i*4+4, mem+i*4+8);
};

// Debug pretty print of a slice of the memory state
void print(int *S, int len){
    for (int j=0; j<4; ++j){
        for (int i=j; i<len*4; i+=4){
                 if (!j && S[i]<-1) printf("DD|");
            else if (!j && S[i]==1) printf("%%%%|");
            else if (S[i]>0) printf("%02x>", S[i]%256);
            else if (S[i]<0) printf("%02x<", (-S[i])%256);
            else if (j<3) printf("  |");
            else printf("__|");
        };
        printf("\n");
    };
};
