#include "hip/hip_runtime.h"
#include <stdio.h>
#include "test_inputs.cu"
#include "rewriter.cu"
#include "now.cu"

const int ints_per_node = 4;

const int threads_per_block = 32;
const int nodes_per_thread = 32;
const int thread_ticks = 1;
const int block_ticks = 16;
const int kernel_calls = 1300;

const int ints_per_thread = nodes_per_thread * ints_per_node;
const int nodes_per_block = nodes_per_thread * threads_per_block;
const int ints_per_block = nodes_per_block * ints_per_node;

__global__ void process(int *mem, int phase){
    __shared__ int smem[ints_per_block];

    int thread_global_num = blockIdx.x*blockDim.x + threadIdx.x;
    int thread_local_num = threadIdx.x;

    // Reads from global to local memory
    int offset = phase ? ints_per_block/2 : 0;
    int smem_idx = thread_local_num * ints_per_thread;
    int mem_idx = thread_global_num * ints_per_thread + offset;

    // printf("KERNEL ph:%d | local = %d/%d, global = %d | mem from %d til %d, smem from %d til %d, smem_size = %d\n", 
        // phase,
        // thread_local_num, 
        // blockDim.x, 
        // thread_global_num,
        // mem_idx,
        // mem_idx+ints_per_thread,
        // smem_idx,
        // smem_idx+ints_per_thread,
        // ints_per_block);

    for (int i=0; i<ints_per_thread; ++i){
        // printf("%d + %d = %d\n", mem_idx, i, mem_idx+i);
        smem[smem_idx+i] = mem_idx+i < memory_ints ? mem[mem_idx+i] : 0;
    }
    __syncthreads();

    // Calls the kernel
    for (int u = 0, ph = 0; u < block_ticks; ++u, ph = !ph){
        int local_offset = ph ? ints_per_thread/2 : 0;
        // int local_offset = 0;
        for (int t = 0; t<thread_ticks; ++t){ // Many ticks on thread's space
            if (ph == 0 || threadIdx.x < blockDim.x-1) {
                // printf("TICK %d %d | mem from %d til %d, smem from %d til %d\n", 
                    // t,
                    // u,
                    // mem_idx+local_offset,
                    // mem_idx+local_offset+ints_per_thread,
                    // smem_idx+local_offset,
                    // smem_idx+local_offset+ints_per_thread);
                tick(smem + smem_idx + local_offset, nodes_per_thread);
            };
            __syncthreads();
        };
    };

    // Writes to global mamory
    for (int i=0; i<ints_per_thread; ++i)
        if (mem_idx+i < memory_ints)
            mem[mem_idx+i] = smem[smem_idx+i];
};

int main(){

    for (int i=0; i < memory_ints; ++i)
        memory[i] = i < program_nodes*4 ? program[i] : 0;

    int *device_memory;
    hipMalloc((void**)&device_memory, sizeof memory);
    hipMemcpy(device_memory, memory, memory_size, hipMemcpyHostToDevice);

    dim3 block_size(threads_per_block, 1);
    dim3 grid_size_0(memory_nodes / nodes_per_block + 1, 1);
    dim3 grid_size_1(memory_nodes / nodes_per_block, 1);
    printf("%d %d \n", block_size.x, grid_size_0.x);

    print(memory, 40);

    double t = now();
    for (int k=0; k<kernel_calls; ++k){
        process<<<grid_size_0, block_size>>>(device_memory, 0);
        process<<<grid_size_1, block_size>>>(device_memory, 1);

        // hipMemcpy(memory, device_memory, memory_size, hipMemcpyDeviceToHost);
        // print(memory, memory_nodes);
    };
    printf("%f\n", now()-t);

    hipMemcpy(memory, device_memory, memory_size, hipMemcpyDeviceToHost);
    print(memory, 40);
    hipFree(device_memory);
}
